#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "vec_math.h"
#include "psc_aabb.cuh"
#include "psc_math.cuh"


#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include "gen_vectorized_visibility.h"

#define	G_PI 3.14159265358979323846f

__device__ float flt_epslion = 1.192092896e-07F;
__device__ float flt_max = 3.402823466e+38F;


#define f3abs(a) ( make_float3(fabsf(a.x),fabsf(a.y),fabsf(a.z)) )
//template<typename T> 
//__device__  float3 f3abs( T a )
//{
//  return make_float3( fabsf(a.x), fabsf(a.y), fabsf(a.z) );
//}
__device__ float length2(const float3& v)
{
  return dot(v, v);
}


__device__ void get_coordinate_system(const cups &ps, float3 &ax, float3 &ay, float3 &az)
{
  ay = ps.vvn;
  ax = make_float3(1,0,0);

  //if( fabsf(dot(ax,ay))>.999999 )
  //  ax = make_float3(0,1,0);
  //az = normalize( cross(ax,ay) );

  float3 ref = lerp( make_float3(ay.z, 0.f, -ay.x), ax, abs(ay.y) );
  az = normalize( cross(ref,ay) );

  ax = normalize( cross(ay,az) );
}

__host__ void my_check_cuda();
#include "cu_initialize_ps.inl"


__device__ float3 es_to_float3( float es, const float3 &ev0, const float3 &ev1  )
{
  return es*(ev1-ev0)+(ev1+ev0)/2; 
}
//__device__ float3 es_to_float3( const CUParam &param, float es, int ei )
//{
//  float3 ev0 = param.vmap[param.emap[ei].v0];
//  float3 ev1 = param.vmap[param.emap[ei].v1];
//  return es_to_float3(es, ev0, ev1);
//}

__device__ float LsPlXRatio( float3 v0, float3 fn, float3 ev0, float3 ev1 )
{
  return -dot((ev1+ev0)/2-v0,fn)/dot(ev1-ev0,fn);
}

__device__ float LsPsXRatio( const CUParam &param, int ei0, const float3 &ps_origin, const float3 &ps_ev0, const float3 &ps_ev1 )
{
  float3 au, av, fn;
  au = param.vmap[param.emap[ei0].v0];
  av = param.vmap[param.emap[ei0].v1];
  fn = cross(au - ps_origin, av - ps_origin);
  return -dot((ps_ev1+ps_ev0)/2-ps_origin,fn)/dot(ps_ev1-ps_ev0,fn);
}




__device__ PointI extrude_endpointX( float3 e0, float3 e1, float3 ax, float3 ay, float3 az )
{
  float3 un = normalize(e1-e0);
  PointI pp;
  pp.ei = -1;
  pp.es = acosf( clamp( dot(un,az), -1.f, 1.f ));
  if(dot(un,ax)<0.f)
    pp.es = -pp.es;
  return pp;
}
__device__ PointI getHemiPointI_ev0( const cups &ps, float3 ax, float3 ay, float3 az )
{
  return extrude_endpointX(ps.origin, ps.ev0, ax,ay,az );
}
__device__ PointI getHemiPointI_ev1( const cups &ps, float3 ax, float3 ay, float3 az )
{
  return extrude_endpointX(ps.origin, ps.ev1, ax,ay,az );
}
__device__ PointI getHemiPointI_es( const cups &ps, float es0, float3 ax, float3 ay, float3 az
){
  float3 u1 = es0*(ps.ev1-ps.ev0) + (ps.ev0+ps.ev1)/2;
  return extrude_endpointX(ps.origin, u1, ax, ay, az );
}




//__device__ PointI extrude_endpoint( const CUParam &param, float3 e0, PointI pp, float3 ax, float3 ay, float3 az )
//{
//  float3 ev0 = param.vmap[ param.emap[pp.ei].v0 ];
//  float3 ev1 = param.vmap[ param.emap[pp.ei].v1 ];
//  float3 e1 = pp.es*(ev1-ev0) + (ev0 + ev1)/2;
//  return extrude_endpointX(e0, e1, ax, ay, az );
//}
//__device__ PointI extrude_endpoint( const CUParam &param, PointI pp, float3 e1, float3 ax, float3 ay, float3 az )
//{
//  float3 ev0 = param.vmap[ param.emap[pp.ei].v0 ];
//  float3 ev1 = param.vmap[ param.emap[pp.ei].v1 ];
//  float3 e0 = pp.es*(ev1-ev0) + (ev0 + ev1)/2;
//  return extrude_endpoint(e0, e1, ax, ay, az );
//}
//__device__ float3 extrude_endpoint( float3 e0, float3 e1, float pos )
//{
//  return normalize(e1-e0)*pos + e0;
//}





/////////////////////////////////////////////////////////
//

__device__ bool ps_edge_intersectxa(const CUParam &param, const float3 &vvn, const float3 &ps_origin, const float3 &ps_ev0, const float3 &ps_ev1, int ei0, int ei1,
  
  int ei1v2, 
  float3 &pt,
  float &es1
)
{
  const float3 &pev0 = ps_ev0;
  const float3 &pev1 = ps_ev1;
  const float3 &pt_ev0 = param.vmap[param.emap[ei1].v0];
  const float3 &pt_ev1 = param.vmap[param.emap[ei1].v1];

  //return ps_edge_intersect(ps_origin, pev0, pev1, pt_ev0, pt_ev1, out );

  float3 au = pev0 - ps_origin;
  float3 av = pev1 - ps_origin;
  float3 fn = (cross(au,av) );
  float3 e0 = pt_ev0 - ps_origin;
  float3 e1 = pt_ev1 - ps_origin;
  float3 ee = pt_ev1 - pt_ev0;
  if( fabsf(dot(ee,fn))<flt_epslion || dot(e0,fn)>0 == dot(e1,fn)>0 )
    return false;

  es1 = -dot((pt_ev1+pt_ev0)/2-ps_origin,fn)/(dot(ee,fn));
  es1 = clamp(es1,-.5f,.5f);
  pt = es1*ee + (pt_ev1+pt_ev0)/2;
  // float3 pt = -dot(e0,fn)/dot(ee,fn)*ee + pt_ev0;
  float3 qt = pt - ps_origin;

  if(dot(cross(au,qt),fn)>0 && dot(cross(qt,av),fn)>0 )
  {
    return true;
  }else
    return false;
}



__device__ bool ps_aabb_cone_intersect(const cubvh *ccc, float3 vertex0, float3 vertex1, float3 vertex2, float3 vvn)
{
  float p0, p1, r;
  float3 m = ccc->m;
  float3 n = ccc->n;
  float3 aabb_center = (m + n)/2;
  float3 extents = (m - n)/2;//m - aabb_center;

  float3 v0 = vertex0 - aabb_center;

  float3 f0 = normalize(vertex1 - vertex0);
  float3 f2 =  - normalize(vertex2 - vertex0);

  float3 a00 = make_float3(0, -f0.z, f0.y);// (1 0 0)xf0
  float3 a02 = make_float3(0, -f2.z, f2.y);// (0 1 0)xf0
  float3 a10 = make_float3(f0.z, 0, -f0.x);// (0 0 1)xf0
  float3 a12 = make_float3(f2.z, 0, -f2.x);// (1 0 0)xf2
  float3 a20 = make_float3(-f0.y, f0.x, 0);// (0 1 0)xf2
  float3 a22 = make_float3(-f2.y, f2.x, 0);// (0 0 1)xf2

  // Test axis a00
  p0 = dot(v0, a00);
  p1 = -dot(f2, a00);
  r = extents.y * fabs(f0.z) + extents.z * fabs(f0.y);

  if( (p1<0?-p0:p0) > r )
  {
    return false;
  }

  // Test axis a02
  p0 = dot(v0, a02);
  p1 =  dot(f0, a02);
  r = extents.y * fabs(f2.z) + extents.z * fabs(f2.y);

  if( (p1<0?-p0:p0) > r )
  {
    return false;
  }

  // Test axis a10
  p0 = dot(v0, a10);
  p1 = -dot(f2, a10);
  r = extents.x * fabs(f0.z) + extents.z * fabs(f0.x);
  if( (p1<0?-p0:p0) > r )
  {
    return false;
  }


  // Test axis a12
  p0 = dot(v0, a12);
  p1 =  dot(f0, a12);
  r = extents.x * fabs(f2.z) + extents.z * fabs(f2.x);

  if( (p1<0?-p0:p0) > r )
  {
    return false;
  }

  // Test axis a20
  p0 = dot(v0, a20);
  p1 = -dot(f2, a20);
  r = extents.x * fabs(f0.y) + extents.y * fabs(f0.x);

  if( (p1<0?-p0:p0) > r )
  {
    return false;
  }


  // Test axis a22
  p0 = dot(v0, a22);
  p1 =  dot(f0, a22);
  r = extents.x * fabs(f2.y) + extents.y * fabs(f2.x);

  if( (p1<0?-p0:p0) > r )
  {
    return false;
  }

  float3 fn = normalize(cross(f2, f0));
  if( fabsf(dot(fn,v0)) > dot(extents,f3abs(fn)) )
    return false;

  //{
  //  float r0 = fmax(extents.x,fmax(extents.y,extents.z))*1.74f * 2;//length(extents); 
  //  float l = length(v0);
  //  if(l>r0)
  //  {
  //    if( dot(v0,(ccc->conec_down))/l > ccc->cos_coner_down )
  //      return false;
  //    if( dot(v0,(ccc->conec_up))/l > ccc->cos_coner_up )
  //      return false;
  //  }
  //}

  float3 v1 = v0 + f0*10;
  float3 v2 = v0 - f2*10;
  if (fmax(v0.x, fmax(v1.x, v2.x)) < -extents.x || fmin(v0.x, fmin(v1.x, v2.x)) > extents.x)
    return false;
  if (fmax(v0.y, fmax(v1.y, v2.y)) < -extents.y || fmin(v0.y, fmin(v1.y, v2.y)) > extents.y)
    return false;
  if (fmax(v0.z, fmax(v1.z, v2.z)) < -extents.z || fmin(v0.z, fmin(v1.z, v2.z)) > extents.z)
    return false;

  return true;
}


__device__ bool ray_aabb_intersect(float3 m, float3 n, curay r)
{
  float3 rpos = r.u0;
  float3 rdir = r.du;
  float t[10];
  t[1] = (n.x - rpos.x) / rdir.x;
  t[2] = (m.x - rpos.x) / rdir.x;
  t[3] = (n.y - rpos.y) / rdir.y;
  t[4] = (m.y - rpos.y) / rdir.y;
  t[5] = (n.z - rpos.z) / rdir.z;
  t[6] = (m.z - rpos.z) / rdir.z;
  t[7] = fmax(fmax(fmin(t[1], t[2]), fmin(t[3], t[4])), fmin(t[5], t[6]));
  t[8] = fmin(fmin(fmax(t[1], t[2]), fmax(t[3], t[4])), fmax(t[5], t[6]));
  t[9] = (t[8] < 0 || t[7] > t[8]) ? 0 : t[7];
  return t[9];
}

__device__ bool ray_triangle_intersect( curay r, float3 v0, float3 v1, float3 v2, float3 &pt )
{
  const float EPSILON = 0.0000001f;
  float3 edge1, edge2, h, s, q;
  float a, f, u, v;
  edge1 = v1 - v0;
  edge2 = v2 - v0;
  h = cross( r.du, edge2);
  a = dot( edge1, h);
  if (a > -EPSILON && a < EPSILON)
    return false;    // This ray is parallel to this triangle.
  f = 1.0 / a;
  s = r.u0 - v0;
  u = f * dot(s,h);
  if (u < 0.0 || u > 1.0)
    return false;
  q = cross(s,edge1);
  v = f * dot(r.du,q);
  if (v < 0.0 || u + v > 1.0)
    return false;
 // At this stage we can compute t to find out where the intersection point is on the line.
  float t = f * dot(edge2,q);
  if (t > EPSILON && t < 1 / EPSILON) // ray intersection
  {
    pt = r.u0 + r.du * t;
   return true;
  }
  else // This means that there is a line intersection but not a ray intersection.
    return false;
}

__device__ bool ray_triangle_intersect( const CUParam &param, int ti, curay r, float3 &pt )
{
  float3 v0 = param.vmap[ param.vimap[3*ti+0] ];
  float3 v1 = param.vmap[ param.vimap[3*ti+1] ];
  float3 v2 = param.vmap[ param.vimap[3*ti+2] ];
  return ray_triangle_intersect( r, v0, v1, v2, pt );
}
//
/////////////////////////////////////////////////////////




__device__ int get_u0_facing( const CUParam &param, const cups &ps, const float3 &pt, int ti, int s, int intersect_ei)
{
  float3 pn0 = normalize(cross(ps.ev0 - ps.origin, ps.ev1 - ps.origin));
  float3 pn1 = normalize(pt - ps.origin);
  float3 pn2 = normalize(cross(pn1, pn0));
  float3 pt1;
  {
    float3 ev0, ev1, ev2;
    ev0 = param.vmap[param.vimap[3 * ti + s]];
    ev1 = param.vmap[param.vimap[3 * ti + (s + 1) % 3]];
    ev2 = param.vmap[param.vimap[3 * ti + (s + 2) % 3]];
    float3 e0 = ev2 - ps.origin;
    float3 e1 = ev1 - ps.origin;
    float3 ee;
    if (dot(e0, pn0) > 0 != dot(e1, pn0) > 0)
      ee = ev1 - ev2;
    else
      ee = ev0 - ev2;
    pt1 = -dot(e0, pn0) / dot(ee, pn0)*ee + ev2;
  }
  bool u0_facing = dot(pt1 - pt, pn2) > 0;
  //return u0_facing;

  int occlusion_count;
  if( param.emap[intersect_ei].t1==-1 )
  {
    if (u0_facing)
      occlusion_count = -1;
    else
      occlusion_count = 1;
  }
  else
  {
    if (u0_facing)
      occlusion_count = -2;
    else
      occlusion_count = 2;
  }

  return occlusion_count;

}

////////////////////////////////////////////////////////
// Line Sample BVH traversal functions
//
__device__ void process_ps_tri_edge_cone(const CUParam &param, cubvh *ccc, const cups &ps
  , EdgeJ *edgej, int &n_edgej
)
{

  int *idx = ccc->idx;
  int ei0 = ps.eidx;

  // for( int i=0; i<ccc->ni; i++ )
  int i=0;
  { 
    int ti = idx[i];
    float3 v0 = param.vmap[ param.vimap[3*ti+0] ];
    float3 n0 = param.nmap[ ti ];

    if (ti != ps.tri_idx)
    {
      int s;
      for (s = 0; s < 3; s++)
      {
        int intersect_ei = param.eimap[3*ti+s];
      
        if( !(dot(n0, ps.origin-v0)>0) && param.emap[intersect_ei].t1!=-1 )
          continue;

        int eivi0, eivi1, ei0vi0, ei0vi1;
        eivi0 = param.emap[intersect_ei].v0;
        eivi1 = param.emap[intersect_ei].v1;
        
        if(ei0!=-1)
        {
          ei0vi0 = param.emap[ei0].v0;
          ei0vi1 = param.emap[ei0].v1;
          if( eivi0 == ei0vi0 || eivi0 == ei0vi1 || eivi1 == ei0vi0 || eivi1 == ei0vi1 )
            continue;
        }

        float3 ev0, en0, en1;
        ev0 = param.vmap[eivi0];
        en0 = param.nmap[param.emap[intersect_ei].t0];

        if(param.emap[intersect_ei].t1!=-1)
          en1 = param.nmap[param.emap[intersect_ei].t1];
        else
          en1 = -en0;

        float3 vv0 = ps.origin - ev0;
        float f0 = dot(vv0, en0);
        float f1 = dot(vv0, en1);

        if( f0>0!=f1>0 )      
        {          
          float es1;
          float3 pt;
          if(
            ps_edge_intersectxa(
              param, ps.vvn, ps.origin, ps.ev0, ps.ev1, ei0, intersect_ei, param.vimap[3*ti+(s+2)%3],
              pt, es1
            )
          ){
            if(n_edgej<INTERSECT_EDGE_SIZE)
            {
              int u0_facing = get_u0_facing(param, ps, pt, ti, s, intersect_ei );
              float es0 = LsPsXRatio(param, intersect_ei, ps.origin, ps.ev0, ps.ev1 );
              es0 = clamp( es0,-.5f,.5f);
              edgej[n_edgej].es0 = es0;
              edgej[n_edgej].u0_facing = u0_facing;
              edgej[n_edgej].es1  = es1;
              edgej[n_edgej].ei1 = intersect_ei;
              n_edgej++;
            }
          }

        }
      }
    }
  }
}

__device__ void ps_bvh_flattened_cone( const CUParam &param, const cups &ps, int addr,
  EdgeJ *edgej, int &n_edgej
){
  cubvh *ccc = (cubvh*) (param.cbvh + addr);
  int my_note[PS_BVH_FLATTENED_CONE_NOTE_SIZE];
  int n = 0;
  my_note[n++] = addr;
  while(n && n<PS_BVH_FLATTENED_CONE_NOTE_SIZE-1 )
  {
    ccc = (cubvh*) (param.cbvh + my_note[n-1]); 
    n--;

    if(ps_aabb_cone_intersect(ccc, ps.origin, ps.ev0, ps.ev1, ps.vvn))
    {
      if(ccc->ni)
        process_ps_tri_edge_cone( param, ccc, ps, edgej, n_edgej );
      else
      {
        if(ccc->left)
          my_note[n++] = ccc->left;
        if(ccc->right)
          my_note[n++] = ccc->right;
      }
    }
  }
}
//
/////////////////////////////////////////////////////////



////////////////////////////////////////////////////////
// Probing Ray BVH traversal functions 
//
__device__ void process_nu0( const CUParam &param, cubvh *ccc, curay r, 
  float3 ps_origin, int ei0, int tri_idx, int &fnu0, int &bnu0 )
{      
  int *idx = ccc->idx;
  for( int i=0; i<ccc->ni; i++ )
  { 
    int ti = idx[i];
     float3 v0 = param.vmap[param.vimap[3 * ti + 0]];
     float3 n0 = param.nmap[ti];
    
    if( ei0==-1 || (param.emap[ei0].t0 != ti && param.emap[ei0].t1 != ti) )
    {
      float3 pp;
      if(ray_triangle_intersect(param, idx[i], r, pp ))
      {
        float d = dot(n0, ps_origin-v0);
        if( d>-flt_epslion )
          fnu0 = fnu0 + 1;
        if( d<+flt_epslion )
          bnu0 = bnu0 + 1;
      }
    }
  }
}

__device__ void rt_cal_nu0_flattened(const CUParam &param, curay r, int addr, 
  float3 ps_origin, int ei0, int tri_idx,  int &fnu0, int &bnu0)
{
  cubvh *ccc = (cubvh*) (param.cbvh + addr);

  int my_note[32];
  int n = 0;

  my_note[n++] = addr;

  while(n && n<31 )
  {
    ccc = (cubvh*) (param.cbvh + my_note[n-1]); 
    n--;

    if(ray_aabb_intersect(ccc->m, ccc->n, r))
    {
      if(ccc->ni)
        process_nu0( param, ccc, r, ps_origin, ei0, tri_idx, fnu0, bnu0);
      if (ccc->left)
        my_note[n++] = ccc->left;
      if (ccc->right)
        my_note[n++] = ccc->right;
    }
  }
}
//
/////////////////////////////////////////////////////////




////////////////////////////////////////////////////////
// PointI functions 
//
//__device__ PointI getPointI( const CUParam &param, int ei, float3 pt )
//{
//  float3 ev0 = param.vmap[ param.emap[ei].v0 ];
//  float3 ev1 = param.vmap[ param.emap[ei].v1 ];
//  PointI pp;
//  pp.ei = ei;
//  pp.es = length(pt-ev0)/length(ev1-ev0)-.5;
//  return pp;
//}
//__device__ PointI getPointI( const CUParam &param, const cups &ps, float theta_hp )
//{
//  PointI pp;
//  pp.ei = ps.eidx;
//  pp.es = theta_hp;
//  if( ps.tri_idx != param.emap[ps.eidx].t0 )
//    pp.es = -pp.es;
//  return pp;
//}
//__device__ PointI getPointI_ev0( const CUParam &param, const cups &ps )
//{
//  return getPointI(param,ps,-.5f);
//}
//__device__ PointI getPointI_ev1( const CUParam &param, const cups &ps )
//{
//  return getPointI(param,ps, .5f);
//}

__device__ float3 eval( const CUParam &param, const PointI &pp, const float3 &origin, const float3 &ax, const float3 &ay, const float3 &az )
{
  if(pp.ei!=-1)
  {
    float3 ev0 = param.vmap[param.emap[pp.ei].v0];
    float3 ev1 = param.vmap[param.emap[pp.ei].v1];
    return pp.es*(ev1-ev0)+(ev1+ev0)/2; 
  }else
  {
    return origin + (cosf(pp.es)*az + sinf(pp.es)*ax)*2;
  }
}


__device__ void extend_memory_record_edge_PointI( int &recount, int n, PointI* &record_edge0, PointI* &record_edge )
{
  if( recount+n>(RECORD_EDGE_SIZE-2) )
  {
    //int pos = atomicAdd( extended, 32 );
    //float3 *tmp = &((float3*)&extended[1])[pos];
    //((int*)record_edge0)[0] = recount;
    //((int*)record_edge0)[1] = pos;

    printf( "record_edge0 memory exceeded\n" );

    //record_edge0 = tmp;
    ((int*)record_edge0)[0] = 0;
    ((int*)record_edge0)[1] = -1;
    record_edge = record_edge0+2;
    recount = 0;
  }
}
__device__ void record_visible_edge(const CUParam &param, const cups &ps, EdgeJ *edgej, int einum, int n, PointI *record_edge00)
{
  PointI *record_edge;
  PointI *record_edge0 = record_edge00;
  ((int*)record_edge0)[0] = 0;
  ((int*)record_edge0)[1] = -1;
  record_edge = record_edge0+2;
  int &recount = *((int*)record_edge00) = 0;

  float3 ax, ay, az;
  PointI u0, u1, u2;
  int i;
  float tu, tv;
  float hp, t0, t1;
  bool A, B, eeflip;

  eeflip = ps.tri_idx != param.emap[ps.eidx].t0;
  A = dot(ps.ev0-ps.origin, ps.vvn)>0;
  B = dot(ps.ev1-ps.origin, ps.vvn)>0;
  hp = LsPlXRatio(ps.origin, ps.vvn, ps.ev0, ps.ev1);
  t0 = tu = (!A && B) ? hp : -.5f;
  t1 = (A && !B) ? hp : .5f;

  for( i=0; i<einum; i++ )
  {
    tv = edgej[i].es0;           
    if(t0<tv)
      break;
    n += edgej[i].u0_facing;
  }

  for(    ; i<einum; i++ )
  {
    tv = edgej[i].es0;           
    if(t1<tv)
      break;
    if( n==0 && fabsf(tv-tu)>.00002f )
    {
      u0.ei = ps.eidx;
      u0.es = eeflip ? -tu : tu;
      u1.ei = ps.eidx;
      u1.es = eeflip ? -tv : tv;
      u2.ei = edgej[i].ei1;
      u2.es = edgej[i].es1;
      extend_memory_record_edge_PointI( recount, 4, record_edge0, record_edge );
      record_edge[recount++] = u0;
      record_edge[recount++] = u1;
      record_edge[recount++] = u1;
      record_edge[recount++] = u2;
    }
    n += edgej[i].u0_facing;
    tu = tv;
  }

  tv = t1;
  if( n==0 && fabsf(tv-tu)>.00002f )
  {
    u0.ei = ps.eidx;
    u0.es = eeflip ? -tu : tu;
    u1.ei = ps.eidx;
    u1.es = eeflip ? -tv : tv;
    extend_memory_record_edge_PointI( recount, 2, record_edge0, record_edge );
    record_edge[recount++] = u0;
    record_edge[recount++] = u1;
    if( A && !B )
    {
      get_coordinate_system(ps, ax, ay, az);
      u2 = getHemiPointI_es( ps, tv, ax,ay,az );
      extend_memory_record_edge_PointI( recount, 2, record_edge0, record_edge );
      record_edge[recount++] = u1;
      record_edge[recount++] = u2;
    }
  }
}
__device__ void record_visible_edge_plane(const CUParam &param, const cups &ps, EdgeJ *edgej, int einum, int n, PointI *record_edge00)
{
  PointI *record_edge;
  PointI *record_edge0 = record_edge00;
  ((int*)record_edge0)[0] = 0;
  ((int*)record_edge0)[1] = -1;
  record_edge = record_edge0+2;
  int &recount = *((int*)record_edge0) = 0;

  float3 ax, ay, az;
  PointI u0, u1, u2;
  int i;
  float tu, tv;

  get_coordinate_system(ps, ax, ay, az);
  tu = -.5f;
  for( i=0; i<einum; i++ )
  {
    tv = edgej[i].es0;
    if( n==0 && fabsf(tv-tu)>.00002f )
    {
      extend_memory_record_edge_PointI( recount, 4, record_edge0, record_edge );
      u0 = getHemiPointI_es( ps,tu, ax,ay,az );
      u1 = getHemiPointI_es( ps,tv, ax,ay,az );
      u2.ei = edgej[i].ei1;
      u2.es = edgej[i].es1;
      record_edge[recount++] = u0;
      record_edge[recount++] = u1;
      record_edge[recount++] = u1;
      record_edge[recount++] = u2;
    }
    n += edgej[i].u0_facing;
    tu = tv;
  }
  tv = .5f;
  if( n==0 && fabsf(tv-tu)>.00002f )
  {
    extend_memory_record_edge_PointI( recount, 2, record_edge0, record_edge );
    u0 = getHemiPointI_es( ps,tu, ax,ay,az );
    u1 = getHemiPointI_es( ps,tv, ax,ay,az );
    record_edge[recount++] = u0;
    record_edge[recount++] = u1;
  }
  
  //*((int*)record_edge0) = recount;
}




__device__ void find_pst_visible_region(const CUParam &param, const cups &ps 
  , PointI *CURecord_vis, EdgeJ *edgej, int &n_edgej)
{
  if(n_edgej>INTERSECT_EDGE_SIZE)
  {
    n_edgej = INTERSECT_EDGE_SIZE;
    printf( "find_pst_visible_region(), buffer overflowed, %s.\n", "theta[]" );
  }
  cg_quick_sort(edgej, n_edgej);

  float thetam;
  {
    float md, d;
    float tu, tv;
    float pu, pv;
    tu = -.5f; //0;
    md = -flt_max;
    for( int i=0; i<n_edgej; i++ )
    {
      tv = edgej[i].es0;
      d  = tv - tu;

      if( md<d )
      {
        md = d;
        pu = tu;
        pv = tv;
      }
      tu = tv;
    }
    tv = .5f;//1;
    d  = tv - tu;
    if( md<d )
    {
      md = d;
      pu = tu;
      pv = tv;
    }
    thetam = (pu+pv)/2.f;
  }
  //thetam = 0;

  int nu0;
  {
    float3 vertexm;
      //vertexm = (ps.ev1-ps.ev0)*(thetam+.5f)+ps.ev0;
      vertexm = es_to_float3( thetam, ps.ev0, ps.ev1 );
      //vertexm = es_to_float3( 0.f, ps.ev0, ps.ev1 );

    curay rnu0;
    int fnu0, bnu0;
      rnu0.u0 = ps.origin;
      rnu0.du = normalize(vertexm - ps.origin);
      fnu0 = 0;
      bnu0 = 0;
    rt_cal_nu0_flattened(param, rnu0, 0, ps.origin, ps.eidx, ps.tri_idx, fnu0, bnu0);
    nu0 = fnu0 + bnu0;
   }

  int n = 0;
  for( int i=0; i<n_edgej && edgej[i].es0<thetam; i++ )
    n += edgej[i].u0_facing;
  n = nu0 - n;

  if( ps.eidx!=-1 )
    record_visible_edge(param, ps, edgej, n_edgej, n, CURecord_vis);
  else
    record_visible_edge_plane(param, ps, edgej, n_edgej, n, CURecord_vis);

}



__global__ void call_ps_bvh_flattened_cone_speed3( CUParam param, cups *ps,  PointI *CURecord_vis, int nps, int ps0)
{
  // gridDim.x 256; [0,(nps+(256-1))/256) <= blockIdx.x;
  // blockDim.x 256; [0,256) <= threadIdx.x;
  int psi = blockIdx.x * blockDim.x + threadIdx.x + ps0;
  if( psi>=nps )
    return;

  EdgeJ edgej[INTERSECT_EDGE_SIZE];
  int n_edgej = 0;

  ps_bvh_flattened_cone(param, ps[psi], 0, edgej, n_edgej);
  find_pst_visible_region(param, ps[psi], &CURecord_vis[psi*RECORD_EDGE_SIZE], edgej, n_edgej);
}


__global__ void tightpack_vis_cal_offset(int *vis_offset, PointI *vis_buf, int nps)
{
  
  // blockDim.x 256;  [0,  32) <- threadIdx.x
  // gridDim.x (nps+(256-1))/256;  [0, (nps+(256-1))/256) <- blockIdx.x

  int id = blockIdx.x*blockDim.x + threadIdx.x;
  if(id>=nps)
    return;

  vis_offset[id] = *((int*)&vis_buf[id*RECORD_EDGE_SIZE]);
}


__global__ void tightpack_vis_copy(PointI *vis_tightpacked, PointI *vis_buf, int *vis_offset, int nps)
{

  // blockDim.x 256;  [0,  32) <- threadIdx.x
  // gridDim.x (nps+(256-1))/256;  [0, (nps+(256-1))/256) <- blockIdx.x
  int id = blockIdx.x*blockDim.x + threadIdx.x;

  if(id>=nps)
    return;

  int ii = vis_offset[id+0];
  int ni = vis_offset[id+1]-vis_offset[id+0];
  PointI *dat = &vis_buf[id*RECORD_EDGE_SIZE]+2;
  for( int i=0; i<ni; i++ )
    vis_tightpacked[ii+i] = dat[i];

}

__global__ void tightpack_vis_cal_vinfo(int *vis_tightpacked_vinfo, int *vis_offset, int *front_back_edge_vinfo, int n_vertex)
{
  // blockDim.x 256;  [0,  32) <- threadIdx.x
  // gridDim.x (n_vertex+1+(256-1))/256;  [0, (n_vertex+1+(256-1))/256) <- blockIdx.x

  int vi = blockIdx.x*blockDim.x + threadIdx.x; 
  if(vi>=n_vertex)
    return;
  vis_tightpacked_vinfo[vi] = vis_offset[   front_back_edge_vinfo[vi] + vi*32   ];
}


__host__ void my_check_cuda()
{
  hipError_t cudaStatus;
  cudaStatus = hipGetLastError();
  if( cudaStatus != hipSuccess )
  {
    printf( "kernel launch error: %s\n", hipGetErrorString(cudaStatus));
    exit(0);
  }
  cudaStatus = hipDeviceSynchronize();
  if( cudaStatus != hipSuccess )
  {
    printf( "cuda sync error: %s\n", hipGetErrorString(cudaStatus));
    exit(0);
  }
}

__host__ void cu_ps_bvh_flattened_cone_speed_g0(
  PointI *&vis_tightpacked, int &vis_tightpacked_size, int &_vis_tightpacked_size, int *vis_tightpacked_vinfo, 
  PointI *vis_buf, int *vis_offset,
  const CUParam &param, cups *ps_buf, int *front_back_edge_vinfo, int nps,
  int n_vertex
){
  //my_check_cuda();

  // depends on ps0 modify call_ps_bvh_flattened_cone_speed3()
  call_ps_bvh_flattened_cone_speed3<<< (nps+(256-1))/256,256 >>>
    (param, ps_buf, vis_buf, nps, 0);
  //printf( "  call_ps_bvh_flattened_cone_speed3 done\n" );
  //my_check_cuda();


  tightpack_vis_cal_offset<<<  (nps+(256-1))/256, 256  >>>(vis_offset+1, vis_buf, nps);
  //printf( "  tightpack_vis_cal_offset done\n" );
  //my_check_cuda();

  thrust::inclusive_scan(thrust::device, vis_offset+1, vis_offset+1 + nps, vis_offset+1);
  //printf( " CUDA thrust::inclusive_scan done\n" );
  //my_check_cuda();

  hipMemcpy( &vis_tightpacked_size, vis_offset+nps, sizeof(int), hipMemcpyDeviceToHost );
  //printf( " CUDA vis_tightpacked_size %i\n", vis_tightpacked_size );
  //my_check_cuda();

  if( _vis_tightpacked_size<vis_tightpacked_size )
  {
    _vis_tightpacked_size = int(vis_tightpacked_size*1.5);
    if(vis_tightpacked)
      hipFree(vis_tightpacked);

    //printf( "  vis_tightpacked %f MB\n", double(_vis_tightpacked_size)*sizeof(PointI)/1024/1024 );
    hipMalloc(&vis_tightpacked, _vis_tightpacked_size*sizeof(PointI));
    //my_check_cuda();
  }

  tightpack_vis_copy<<< (nps+(256-1))/256,256 >>>( vis_tightpacked, vis_buf, vis_offset, nps );
  //my_check_cuda();

  tightpack_vis_cal_vinfo<<< (n_vertex+1+(256-1))/256, 256 >>>(vis_tightpacked_vinfo, vis_offset, front_back_edge_vinfo, n_vertex+1);
  //my_check_cuda();
}





__global__ void ps_bvh_flattened_cone_testing1( CUParam param, cups *ps,  PointI *CURecord_vis, int nps, int ps0)
{
  // gridDim.x 256; [0,(nps+(256-1))/256) <= blockIdx.x;
  // blockDim.x 256; [0,256) <= threadIdx.x;
  int psi = blockIdx.x * blockDim.x + threadIdx.x + ps0;
  if( psi>=nps )
    return;

  EdgeJ edgej[INTERSECT_EDGE_SIZE];
  int n_edgej = 0;

  ps_bvh_flattened_cone(param, ps[psi], 0, edgej, n_edgej);
}

__global__ void ps_bvh_flattened_cone_testing2( CUParam param, cups *psbuf,  PointI *CURecord_vis, int nps, int ps0, int *fnu0_buf, int *bnu0_buf)
{
  // gridDim.x 256; [0,(nps+(256-1))/256) <= blockIdx.x;
  // blockDim.x 256; [0,256) <= threadIdx.x;
  int psi = blockIdx.x * blockDim.x + threadIdx.x + ps0;
  if( psi>=nps )
    return;

  cups ps;
  float3 vertexm;
  curay rnu0;
  int fnu0, bnu0;

  ps = psbuf[psi];
  vertexm = (ps.ev0+ps.ev1)/2;
  rnu0.u0 = ps.origin;
  rnu0.du = normalize(vertexm - ps.origin);
  fnu0 = 0;
  bnu0 = 0;

  rt_cal_nu0_flattened(param, rnu0, 0, ps.origin, ps.eidx, ps.tri_idx, fnu0, bnu0);

  fnu0_buf[psi] = fnu0;
  bnu0_buf[psi] = bnu0;
}


#include <time.h>

__host__ void cu_ps_bvh_flattened_cone_speed_testing(
  PointI *vis_buf, int *vis_offset,
  const CUParam &param, cups *ps_buf, int *front_back_edge_vinfo, int nps,
  int n_vertex, int n_update
){

  int vertex0;
  int ps0, nps_update;

  vertex0  = n_vertex-n_update;
  hipMemcpy( &ps0, front_back_edge_vinfo+vertex0, sizeof(int), hipMemcpyDeviceToHost );
  ps0 = ps0 + 32*vertex0;
  nps_update = nps - ps0;
  //printf( "%i -> %i\n", nps, nps_update );

  int i;
  int nprocess = 100;

  clock_t t0, t1;
  t0 = clock();
  for( i=0; i<nprocess; i++ )
  {
    // depends on ps0 modify call_ps_bvh_flattened_cone_speed3()
    ps_bvh_flattened_cone_testing1<<< (nps_update+(256-1))/256,256 >>>
      (param, ps_buf, vis_buf, nps, ps0);
    hipDeviceSynchronize();
  }
  t1 = clock();
  printf( "Line sample BVH traversal: average processing time per pass %f s\n", float(t1-t0)/CLOCKS_PER_SEC/nprocess );

  int *fnu0_buf, *bnu0_buf;
  hipMalloc( &fnu0_buf, ((n_vertex+(256-1))/256) * 256 * sizeof(int) );
  hipMalloc( &bnu0_buf, ((n_vertex+(256-1))/256) * 256 * sizeof(int) );

  t0 = clock();
  for( i=0; i<nprocess; i++ )
  {
    ps_bvh_flattened_cone_testing2<<< (nps_update+(256-1))/256,256 >>>
      (param, ps_buf, vis_buf, nps, ps0, fnu0_buf, bnu0_buf);
    hipDeviceSynchronize();
  }
  t1 = clock();

  printf( "Probing ray BVH traversal: average processing time per pass %f s\n", float(t1-t0)/CLOCKS_PER_SEC/nprocess );
  printf( "Number of line sample %i\n", nps_update);

  hipFree(fnu0_buf);
  hipFree(bnu0_buf);
  
}





////////////////////////////////////////////////////////
// Not in used functions
//
__device__ void process_node_triangles( const CUParam &param, cubvh *ccc, curay r, CUType0 *res )
{      
  int *idx = ccc->idx;
  for( int i=0; i<ccc->ni; i++ )
  { 
    float3 pt;
    if( ray_triangle_intersect(param, idx[i], r, pt ) )
    {
      if( 
        res->ti<0 || 
        length2(pt-r.u0)<length2(res->pt-r.u0)
      ){
        res->pt = pt;
        res->ti = idx[i];
      }
    }
  }
}
__device__ void rt_bvh_flattened( const CUParam &param, curay r, int addr, CUType0 *res)
{
  cubvh *ccc = (cubvh*) (param.cbvh + addr);

  int my_note[32];
  int n = 0;

  my_note[n++] = addr;

  while(n && n<31 )
  {
    ccc = (cubvh*) (param.cbvh + my_note[n-1]); 
    n--;

    if(ray_aabb_intersect(ccc->m, ccc->n, r))
    {
      if(ccc->ni)
        process_node_triangles( param, ccc, r, res );
      if (ccc->left)
        my_note[n++] = ccc->left;
      if (ccc->right)
        my_note[n++] = ccc->right;
    }
  }
}
__global__ void call_rt_bvh( CUParam param, curay *rays, CUType0 *results )
{
  int i = threadIdx.x;
  results[i].ti = -1;
  rt_bvh_flattened(param, rays[i], 0, &results[i]);
}
__host__ void cu_rt_bvh( const CUParam &param, curay *rays, CUType0 *results, int ns )
{
  call_rt_bvh<<<1,1>>>( param, rays, results);

  hipError_t cudaStatus;

  cudaStatus = hipGetLastError();
  if( cudaStatus != hipSuccess )
    printf( "kernel launch error: %s\n", hipGetErrorString(cudaStatus));

  cudaStatus = hipDeviceSynchronize();
  if( cudaStatus != hipSuccess )
    printf( "cuda sync error: %s\n", hipGetErrorString(cudaStatus));
}
////////////////////////////////////////////////////////
